#include "hip/hip_runtime.h"
//
// Created by qaze on 01.11.2021.
//

#include <util.h>
#include <iostream>
#include "../include/gpu_computation_unit.cuh"
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include <iomanip>

#define BLOCK_SIZE 256
#define EPSILON 0.01

__device__
bool d_finished;  // TODO perf

template<typename T>
GpuComputationUnit<T>::GpuComputationUnit(Grid<T> &grid, Grid<T> &previous, Synchronisation barrier, int chunkStart, int chunkSize, bool leader)
        :ComputationUnit<T>(grid, previous, barrier, chunkStart, chunkSize, leader) {

    int iterations = 0;
    int iterations_print = 1;
    double startTime = timeMs();
    bool h_finished = true;

    int copyBlockCount = (grid.totalSize + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int stepBlockCount = (chunkSize + BLOCK_SIZE - 1) / BLOCK_SIZE;
    std::cout << "copy block count: " << copyBlockCount << std::endl;
    std::cout << "step block count: " << stepBlockCount << std::endl;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    printf("Device name: %s\n", prop.name);


    for (int i = 0; i < 1000; ++i) {
        if (leader) {
            copy<<<copyBlockCount, BLOCK_SIZE>>>(grid.totalSize, grid.raw(), previous.raw());
        }

        hipDeviceSynchronize();
        barrier.synchronise();

        h_finished = true;
        hipMemcpyToSymbol(HIP_SYMBOL(d_finished), &h_finished, sizeof(bool));

        step<<<stepBlockCount, BLOCK_SIZE>>>(chunkStart + chunkSize, grid.raw(), previous.raw(), grid.sizeY, chunkStart, EPSILON);

        iterations++;
        if (iterations == iterations_print) {
            std::cout << "  " << std::setw(8) << iterations << "\n";
            iterations_print = 2 * iterations_print;
        }

        hipMemcpyFromSymbol(&h_finished, HIP_SYMBOL(d_finished), sizeof(bool));

        hipDeviceSynchronize();
        barrier.synchronise();
    }

    std::cout << "total time " << timeMs() - startTime;
}

template<typename T>
__global__ void copy(int n, T *source, T *destination) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        destination[i] = source[i];
    }
}

template<typename T>
__global__ void step(int n, T *current, T *previous, int wrap, int start, double epsilon) { // TODO perf
    int index = start + blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        if (previous[i] != 0.0 && previous[i] != 100.0) { // TODO correctness, perf
            current[i] = (previous[i - 1] + previous[i + 1] + previous[i - wrap] + previous[i + wrap]) / 4.0;
        }
        if (fabs(current[i] - previous[i]) > epsilon) d_finished = false; // TODO perf
    }
}

template<typename T>
GpuComputationUnit<T>::~GpuComputationUnit() {

}

template class GpuComputationUnit<float>;
template class GpuComputationUnit<double>;
